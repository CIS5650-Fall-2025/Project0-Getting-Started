#include "hip/hip_runtime.h"
#include "common.h"

#include <>

#include <cmath>
#include <iostream>
#include <random>

// TODO 10: Implement the matrix multiplication kernel
__global__ void matrixMultiplicationNaive(float* const matrixP, const float* const matrixM, const float* const matrixN,
                                          const unsigned sizeMX, const unsigned sizeNY, const unsigned sizeXY)
{
    // TODO 10a: Compute the P matrix global index for each thread along x and y dimentions.
    // Remember that each thread of the kernel computes the result of 1 unique element of P
    unsigned px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned py = blockIdx.y * blockDim.y + threadIdx.y; 

    // TODO 10b: Check if px or py are out of bounds. If they are, return.
    if (px >= sizeMX || py >= sizeNY) {
        return; 
    }

    // TODO 10c: Compute the dot product for the P element in each thread
    // This loop will be the same as the host loop
    float dot = 0; 
    for (int k = 0; k < sizeXY; ++k) {
        dot += matrixM[k * sizeMX + px] * matrixN[py * sizeXY + k];
    }

    // TODO 10d: Copy dot to P matrix
    // matrixP[] = dot;
    matrixP[py * sizeMX + px] = dot;
}

int main(int argc, char *argv[])
{
    // TODO 1: Initialize sizes. Start with simple like 16x16, then try 32x32.
    // Then try large multiple-block square matrix like 64x64 up to 2048x2048.
    // Then try square, non-power-of-two like 15x15, 33x33, 67x67, 123x123, and 771x771
    // Then try rectangles with powers of two and then non-power-of-two.
    const unsigned sizeMX = 15;
    const unsigned sizeXY = 15;
    const unsigned sizeNY = 33;

    // TODO 2: Allocate host 1D arrays for:
    // matrixM[sizeMX, sizeXY]
    // matrixN[sizeXY, sizeNY]
    // matrixP[sizeMX, sizeNY]
    // matrixPGold[sizeMX, sizeNY]
    float* matrixM = new float[sizeMX * sizeXY];
    float* matrixN = new float[sizeXY * sizeNY]; 
    float* matrixP = new float [sizeMX * sizeNY];
    float* matrixPGold = new float[sizeMX * sizeNY];

    // LOOK: Setup random number generator and fill host arrays and the scalar a.
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    // Fill matrix M on host
    for (unsigned i = 0; i < sizeMX * sizeXY; i++)
        matrixM[i] = dist(mt);

    // Fill matrix N on host
    for (unsigned i = 0; i < sizeXY * sizeNY; i++)
        matrixN[i] = dist(mt);

    // TODO 3: Compute "gold" reference standard
    // for py -> 0 to sizeNY
    //   for px -> 0 to sizeMX
    //     initialize dot product accumulator
    //     for k -> 0 to sizeXY
    //       dot = m[k, px] * n[py, k]
    //  matrixPGold[py, px] = dot

    for (int py = 0; py < sizeNY; ++py) {
        for (int px = 0; px < sizeMX; ++px) {
            float accum = 0; 
            for (int k = 0; k < sizeXY; ++k) {
                accum += matrixM[k * sizeMX + px] * matrixN[py * sizeXY + k]; 
            }
            matrixPGold[py * sizeMX + px] = accum;  
        }
    }

    // Device arrays
    float *d_matrixM, *d_matrixN, *d_matrixP;

    // TODO 4: Allocate memory on the device for d_matrixM, d_matrixN, d_matrixP.
    const size_t sizeBytesM = sizeMX * sizeXY * sizeof(float); 
    const size_t sizeBytesN = sizeXY * sizeNY * sizeof(float); 
    const size_t sizeBytesP = sizeMX * sizeNY * sizeof(float); 
    CUDA(hipMalloc(&d_matrixM, sizeBytesM)); 
    CUDA(hipMalloc(&d_matrixN, sizeBytesN)); 
    CUDA(hipMalloc(&d_matrixP, sizeBytesP)); 

    // TODO 5: Copy array contents of M and N from the host (CPU) to the device (GPU)
    CUDA(hipMemcpy(d_matrixM, matrixM, sizeBytesM, hipMemcpyHostToDevice)); 
    CUDA(hipMemcpy(d_matrixN, matrixN, sizeBytesN, hipMemcpyHostToDevice)); 

    CUDA(hipDeviceSynchronize());

    ////////////////////////////////////////////////////////////
    std::cout << "****************************************************" << std::endl;
    std::cout << "***Matrix Multiplication***" << std::endl;

    // LOOK: Use the clearHostAndDeviceArray function to clear matrixP and d_matrixP
    clearHostAndDeviceArray(matrixP, d_matrixP, sizeMX * sizeNY);

    // TODO 6: Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
    // Calculate number of blocks along X and Y in a 2D CUDA "grid" using divup
    // HINT: The shape of matrices has no impact on launch configuaration
    const int threadCountX = 32; 
    const int threadCountY = 32; 

    DIMS dims;
    dims.dimBlock = dim3(threadCountX, threadCountY, 1);
    dims.dimGrid  = dim3(divup(sizeMX, threadCountX), divup(sizeNY, threadCountY), 1);

    // TODO 7: Launch the matrix transpose kernel
    // matrixMultiplicationNaive<<<>>>();
    matrixMultiplicationNaive << <dims.dimGrid, dims.dimBlock >> > (d_matrixP, d_matrixM, d_matrixN, sizeMX, sizeNY, sizeXY); 
        
    // TODO 8: copy the answer back to the host (CPU) from the device (GPU)
    CUDA(hipMemcpy(matrixP, d_matrixP, sizeBytesP, hipMemcpyDeviceToHost)); 

    // LOOK: Use compareReferenceAndResult to check the result
    compareReferenceAndResult(matrixPGold, matrixP, sizeMX * sizeNY, 1e-3);

    std::cout << "****************************************************" << std::endl << std::endl;
    ////////////////////////////////////////////////////////////

    // TODO 9: free device memory using hipFree
    hipFree(d_matrixM); 
    hipFree(d_matrixN); 
    hipFree(d_matrixP); 

    // free host memory
    delete[] matrixM;
    delete[] matrixN;
    delete[] matrixP;
    delete[] matrixPGold;

    // successful program termination
    return 0;
}
