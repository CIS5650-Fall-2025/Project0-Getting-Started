#include "hip/hip_runtime.h"
#include "common.h"

#include <>

#include <cmath>
#include <iostream>
#include <random>

// TODO 10: Implement the matrix multiplication kernel
__global__ void matrixMultiplicationNaive(float* const matrixP, const float* const matrixM, const float* const matrixN,
                                          const unsigned sizeMX, const unsigned sizeNY, const unsigned sizeXY)
{
    // TODO 10a: Compute the P matrix global index for each thread along x and y dimentions.
    // Remember that each thread of the kernel computes the result of 1 unique element of P
    unsigned px;
    unsigned py;
	px = blockIdx.x * blockDim.x + threadIdx.x;
	py = blockDim.y * blockIdx.y + threadIdx.y;
    // TODO 10b: Check if px or py are out of bounds. If they are, return.
    if(px >= sizeMX || py >= sizeNY)
		return;
    // TODO 10c: Compute the dot product for the P element in each thread
    // This loop will be the same as the host loop
    float dot = 0.0;
    for (unsigned k = 0; k < sizeXY; k++)
		dot += matrixM[k * sizeMX + px] * matrixN[py * sizeXY + k];
    // TODO 10d: Copy dot to P matrix
    // matrixP[] = dot;
	matrixP[py * sizeMX + px] = dot;
}

int main(int argc, char *argv[])
{
    // TODO 1: Initialize sizes. Start with simple like 16x16, then try 32x32.
    // Then try large multiple-block square matrix like 64x64 up to 2048x2048.
    // Then try square, non-power-of-two like 15x15, 33x33, 67x67, 123x123, and 771x771
    // Then try rectangles with powers of two and then non-power-of-two.
    const unsigned sizeMX = 23;
    const unsigned sizeXY = 67;
    const unsigned sizeNY = 45;

    // TODO 2: Allocate host 1D arrays for:
    // matrixM[sizeMX, sizeXY]
    // matrixN[sizeXY, sizeNY]
    // matrixP[sizeMX, sizeNY]
    // matrixPGold[sizeMX, sizeNY]
    float* matrixM;
    float* matrixN;
    float* matrixP;
    float* matrixPGold;

	int size_in_bytes_M = sizeMX * sizeXY * sizeof(float);
	CUDA(hipHostMalloc((void**)&matrixM, size_in_bytes_M));
	int size_in_bytes_N = sizeXY * sizeNY * sizeof(float);
	CUDA(hipHostMalloc((void**)&matrixN, size_in_bytes_N));
	int size_in_bytes_P = sizeMX * sizeNY * sizeof(float);
	CUDA(hipHostMalloc((void**)&matrixP, size_in_bytes_P));
	matrixPGold = new float[sizeMX * sizeNY];
    // LOOK: Setup random number generator and fill host arrays and the scalar a.
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    // Fill matrix M on host
    for (unsigned i = 0; i < sizeMX * sizeXY; i++)
        matrixM[i] = dist(mt);

    // Fill matrix N on host
    for (unsigned i = 0; i < sizeXY * sizeNY; i++)
        matrixN[i] = dist(mt);

    // TODO 3: Compute "gold" reference standard
    // for py -> 0 to sizeNY
    //   for px -> 0 to sizeMX
    //     initialize dot product accumulator
    //     for k -> 0 to sizeXY
    //       dot = m[k, px] * n[py, k]
    //  matrixPGold[py, px] = dot
    for (unsigned py = 0; py < sizeNY; py++)
        for (unsigned px = 0; px < sizeMX; px++)
        {
            float dot = 0.0;
            for (unsigned k = 0; k < sizeXY; k++)
                dot += matrixM[k * sizeMX + px] * matrixN[py * sizeXY + k];
            matrixPGold[py * sizeMX + px] = dot;
		}
    // Device arrays
    float *d_matrixM, *d_matrixN, *d_matrixP;

    // TODO 4: Allocate memory on the device for d_matrixM, d_matrixN, d_matrixP.
	CUDA(hipMalloc((void**)&d_matrixM, size_in_bytes_M));
	CUDA(hipMalloc((void**)&d_matrixN, size_in_bytes_N));
	CUDA(hipMalloc((void**)&d_matrixP, size_in_bytes_P));
    // TODO 5: Copy array contents of M and N from the host (CPU) to the device (GPU)
	CUDA(hipMemcpy(d_matrixM, matrixM, size_in_bytes_M, hipMemcpyHostToDevice));
	CUDA(hipMemcpy(d_matrixN, matrixN, size_in_bytes_N, hipMemcpyHostToDevice));
    CUDA(hipDeviceSynchronize());

    ////////////////////////////////////////////////////////////
    std::cout << "****************************************************" << std::endl;
    std::cout << "***Matrix Multiplication***" << std::endl;

    // LOOK: Use the clearHostAndDeviceArray function to clear matrixP and d_matrixP
    clearHostAndDeviceArray(matrixP, d_matrixP, sizeMX * sizeNY);

    // TODO 6: Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
    // Calculate number of blocks along X and Y in a 2D CUDA "grid" using divup
    // HINT: The shape of matrices has no impact on launch configuaration
    DIMS dims;
    dims.dimBlock = dim3(16, 16, 1);
	dims.dimGrid = dim3(divup(sizeMX, dims.dimBlock.x), divup(sizeNY, dims.dimBlock.y), 1);

    // TODO 7: Launch the matrix transpose kernel
    // matrixMultiplicationNaive<<<>>>();
	matrixMultiplicationNaive <<<dims.dimGrid, dims.dimBlock >>> (d_matrixP, d_matrixM, d_matrixN, sizeMX, sizeNY, sizeXY);
    // TODO 8: copy the answer back to the host (CPU) from the device (GPU)
	CUDA(hipMemcpy(matrixP, d_matrixP, size_in_bytes_P, hipMemcpyDeviceToHost));
    // LOOK: Use compareReferenceAndResult to check the result
    compareReferenceAndResult(matrixPGold, matrixP, sizeMX * sizeNY, 1e-3);

    std::cout << "****************************************************" << std::endl << std::endl;
    ////////////////////////////////////////////////////////////

    // TODO 9: free device memory using hipFree
	CUDA(hipFree(d_matrixM));
	CUDA(hipFree(d_matrixN));
	CUDA(hipFree(d_matrixP));
    // free host memory
	CUDA(hipHostFree(matrixM));
	CUDA(hipHostFree(matrixN));
	CUDA(hipHostFree(matrixP));
    //delete[] matrixM;
    //delete[] matrixN;
    //delete[] matrixP;
    delete[] matrixPGold;

    // successful program termination
    return 0;
}
