#include "hip/hip_runtime.h"
#include "common.h"

#include <>

#include <cmath>
#include <iostream>
#include <random>

// TODO 10: Implement the matrix multiplication kernel
__global__ void matrixMultiplicationNaive(float* const matrixP, const float* const matrixM, const float* const matrixN,
                                          const unsigned sizeMX, const unsigned sizeNY, const unsigned sizeXY)
{
    // TODO 10a: Compute the P matrix global index for each thread along x and y dimentions.
    // Remember that each thread of the kernel computes the result of 1 unique element of P

    // X is the Cols -> N
    // Y is the Rows -> m

    unsigned px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned py = blockIdx.y * blockDim.y + threadIdx.y;

    // TODO 10b: Check if px or py are out of bounds. If they are, return.
    if (px >= sizeMX || py >= sizeNY)
        return;

    // TODO 10c: Compute the dot product for the P element in each thread
    // This loop will be the same as the host loop
    float dot = 0.0;
    for (int k = 0; k < sizeXY; k++) {
        dot += matrixM[px * sizeXY + k] * matrixN[k * sizeXY + py];
    }

    // TODO 10d: Copy dot to P matrix
    // matrixP[] = dot;

    //matrixP[px * sizeMX + py] = dot;
    matrixP[px * sizeNY + py] = dot;
}

int main(int argc, char *argv[])
{
    // TODO 1: Initialize sizes. Start with simple like 16x16, then try 32x32.
    // Then try large multiple-block square matrix like 64x64 up to 2048x2048.
    // Then try square, non-power-of-two like 15x15, 33x33, 67x67, 123x123, and 771x771
    // Then try rectangles with powers of two and then non-power-of-two.
    const unsigned sizeM = 15;
    const unsigned sizeXY = 13;
    const unsigned sizeN = 28;

    // TODO 2: Allocate host 1D arrays for:
    // matrixM[sizeMX, sizeXY]
    // matrixN[sizeXY, sizeNY]
    // matrixP[sizeMX, sizeNY]
    // matrixPGold[sizeMX, sizeNY]
    float* matrixM = new float[sizeM * sizeXY];
    float* matrixN = new float[sizeXY * sizeN];
    float* matrixP = new float[sizeM * sizeN];
    float* matrixPGold = new float[sizeM * sizeN];

    // LOOK: Setup random number generator and fill host arrays and the scalar a.
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    // Fill matrix M on host
    for (unsigned i = 0; i < sizeM * sizeXY; i++)
        matrixM[i] = dist(mt);

    // Fill matrix N on host
    for (unsigned i = 0; i < sizeXY * sizeN; i++)
        matrixN[i] = dist(mt);

    // TODO 3: Compute "gold" reference standard
    // for py -> 0 to sizeNY
    //   for px -> 0 to sizeMX
    //     initialize dot product accumulator
    //     for k -> 0 to sizeXY
    //       dot = m[k, px] * n[py, k]
    //  matrixPGold[py, px] = dot

    for (int i = 0; i < sizeN; i++) {
        for (int j = 0; j < sizeM; j++) {
            float dot = 0.0;
            for (int k = 0; k < sizeXY; k++) {
                dot += matrixM[j * sizeXY + k] * matrixN[k * sizeXY + i];
            }

            matrixPGold[j * sizeN + i] = dot;
        }
    }

    // Device arrays
    float *d_matrixM, *d_matrixN, *d_matrixP;

    // TODO 4: Allocate memory on the device for d_matrixM, d_matrixN, d_matrixP.

    CUDA(hipMalloc((void**)&d_matrixM, sizeM * sizeXY * sizeof(float)));
    CUDA(hipMalloc((void**)&d_matrixN, sizeXY * sizeN * sizeof(float)));
    CUDA(hipMalloc((void**)&d_matrixP, sizeM * sizeN * sizeof(float)));

    // TODO 5: Copy array contents of M and N from the host (CPU) to the device (GPU)

    CUDA(hipMemcpy(d_matrixM, matrixM, sizeM * sizeXY * sizeof(float), hipMemcpyHostToDevice));
    CUDA(hipMemcpy(d_matrixN, matrixN, sizeXY * sizeN * sizeof(float), hipMemcpyHostToDevice));

    CUDA(hipDeviceSynchronize()); 

    ////////////////////////////////////////////////////////////
    std::cout << "****************************************************" << std::endl;
    std::cout << "***Matrix Multiplication***" << std::endl;

    // LOOK: Use the clearHostAndDeviceArray function to clear matrixP and d_matrixP
    clearHostAndDeviceArray(matrixP, d_matrixP, sizeM * sizeN);

    // TODO 6: Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
    // Calculate number of blocks along X and Y in a 2D CUDA "grid" using divup
    // HINT: The shape of matrices has no impact on launch configuaration

    // RESULT MATRIX IS sizeMX by sizeNY

    const unsigned numThreadsM = 16;
    const unsigned numThreadsN = 16;


    const unsigned numBlocksN = divup(sizeM, numThreadsM);
    const unsigned numBlocksM = divup(sizeN, numThreadsN);


    DIMS dims;
    dims.dimBlock = dim3(numThreadsN, numThreadsM, 1);
    dims.dimGrid  = dim3(numBlocksN, numBlocksM, 1);

    // TODO 7: Launch the matrix transpose kernel
    // matrixMultiplicationNaive<<<>>>();

    matrixMultiplicationNaive<<<dims.dimGrid, dims.dimBlock>>>(d_matrixP, d_matrixM, d_matrixN, sizeM, sizeN, sizeXY);

    // TODO 8: copy the answer back to the host (CPU) from the device (GPU)

    CUDA(hipMemcpy(matrixP, d_matrixP, sizeM * sizeN * sizeof(float), hipMemcpyDeviceToHost));

    // PRINTS KERNEL RESULT AND GOLD RESULT FOR DEBUGGING

    std::cout << "KERNEL RESULT:" << std::endl << std::endl;
    printMatrix(matrixP, sizeM, sizeN);
    std::cout << "EXPECTED RESULT:" << std::endl << std::endl;
    printMatrix(matrixPGold, sizeM, sizeN);

    // LOOK: Use compareReferenceAndResult to check the result
    compareReferenceAndResult(matrixPGold, matrixP, sizeM * sizeN, 1e-3);

    std::cout << "****************************************************" << std::endl << std::endl;
    ////////////////////////////////////////////////////////////

    // TODO 9: free device memory using hipFree

    CUDA(hipFree(d_matrixM));
    CUDA(hipFree(d_matrixN));
    CUDA(hipFree(d_matrixP));

    // free host memory
    delete[] matrixM;
    delete[] matrixN;
    delete[] matrixP;
    delete[] matrixPGold;

    // successful program termination
    return 0;
}
