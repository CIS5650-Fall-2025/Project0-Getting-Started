#include "hip/hip_runtime.h"
#include "common.h"

#include <>

#include <cmath>
#include <iostream>
#include <random>

__global__ void saxpy(float* const z, const float* const x, const float* const y, const float a, const unsigned size)
{
    // TODO 9: Compute the global index for each thread.
    unsigned idx = 0;

    // TODO 10: Check if idx is out of bounds. If yes, return.
    if (idx >= 0)
        return;

    // TODO 11: Perform the SAXPY operation: z = a * x + y.
}

int main(int argc, char *argv[])
{
    // Set the size. Start with something simple like 64.
    // Optional: Try out these sizes: 256, 1024, 2048, 14, 103, 1025, 3127
    const unsigned size = 64;

    // Host arrays.
    float* x = new float[size];
    float* y = new float[size];
    float* z = new float[size];

    // LOOK: We use this "gold" array to store the CPU result to be compared with GPU result
    float* z_gold = new float[size];

    // LOOK: Setup random number generator and fill host arrays and the scalar a.
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    // Fill matrix x and y, then a
    for (unsigned i = 0; i < size; i++) {
        x[i] = dist(mt);
        y[i] = dist(mt);
    }
    const float a = dist(mt);

    // Compute "gold" reference standard
    for (unsigned i = 0; i < size; i++)
        z_gold[i] = a * x[i] + y[i];

    // Device arrays
    float *d_x, *d_y, *d_z;
    size_t count = size * sizeof(float);

    // Allocate memory on the device.
    CUDA(hipMalloc((void**) &d_x, count));
    CUDA(hipMalloc((void**) &d_y, count));
    CUDA(hipMalloc((void**) &d_z, count));

    // Copy array contents of X and Y from the host (CPU) to the device (GPU).
    CUDA(hipMemcpy(d_x, x, count, hipMemcpyHostToDevice));
    CUDA(hipMemcpy(d_y, y, count, hipMemcpyHostToDevice));

    CUDA(hipDeviceSynchronize());

    ////////////////////////////////////////////////////////////
    std::cout << "****************************************************" << std::endl;
    std::cout << "***SAXPY***" << std::endl;

    // LOOK: Use the preprocess function to clear z and d_z
    clearHostAndDeviceArray(z, d_z, size);

    // TODO 4: Setup threads and blocks.
    // Start threadPerBlock as 128, then try out differnt configurations: 32, 64, 256, 512, 1024
    // Use divup to get the number of blocks to launch.
    const unsigned threadsPerBlock = 0;

    // TODO 5: Implement the divup function in common.cpp
    const unsigned blocks = divup(size, threadsPerBlock);

    // TODO 6: Launch the GPU kernel with blocks and threadPerBlock as launch configuration
    // saxpy<<< >>> (....);

    // TODO 7: Copy the answer back to the host (CPU) from the device (GPU).
    // Copy what you did in 3, except for d_z -> z.

    // LOOK: Use postprocess to check the result
    compareReferenceAndResult(z_gold, z, size, 1e-6);
    std::cout << "****************************************************" << std::endl << std::endl;
    ////////////////////////////////////////////////////////////

    // TODO 8: free device memory using hipFree
    // CUDA(hipFree(device pointer));

    // free host memory
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] z_gold;

    // successful program termination
    return 0;
}
