#include "hip/hip_runtime.h"
#include "common.h"

#include <>

#include <cmath>
#include <iostream>
#include <random>

__global__ void saxpy(float* const z, const float* const x, const float* const y, const float a, const unsigned size)
{
    // Compute the global index for each thread.
    // Assumptions: blockDim.y == 1 (so it's a N x 1 configuration of blocks), threadIdx.y == 1 (so it's a
    // N x 1 configuration of threads)
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is out of bounds. If yes, return.
    if (idx >= size)
        return;

    // Perform the SAXPY operation: z = a * x + y.
    z[idx] = a * x[idx] + y[idx];
}

int main(int argc, char *argv[])
{
    // Set the size. Start with something simple like 64.
    // Optional: Try out these sizes: 256, 1024, 2048, 14, 103, 1025, 3127
    const unsigned size = 257;

    // Host arrays.
    float* x = new float[size];
    float* y = new float[size];
    float* z = new float[size];

    // LOOK: We use this "gold" array to store the CPU result to be compared with GPU result
    float* z_gold = new float[size];

    // LOOK: Setup random number generator and fill host arrays and the scalar a.
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    // Fill matrix x and y, then a
    for (unsigned i = 0; i < size; i++) {
        x[i] = dist(mt);
        y[i] = dist(mt);
    }
    const float a = dist(mt);

    // Compute "gold" reference standard
    for (unsigned i = 0; i < size; i++)
        z_gold[i] = a * x[i] + y[i];

    // Device arrays
    float *d_x, *d_y, *d_z;
    size_t count = size * sizeof(float);

    // Allocate memory on the device.
    CUDA(hipMalloc((void**) &d_x, count));
    CUDA(hipMalloc((void**) &d_y, count));
    CUDA(hipMalloc((void**) &d_z, count));

    // Copy array contents of X and Y from the host (CPU) to the device (GPU).
    CUDA(hipMemcpy(d_x, x, count, hipMemcpyHostToDevice));
    CUDA(hipMemcpy(d_y, y, count, hipMemcpyHostToDevice));

    CUDA(hipDeviceSynchronize());

    ////////////////////////////////////////////////////////////
    std::cout << "****************************************************" << std::endl;
    std::cout << "***SAXPY***" << std::endl;

    // LOOK: Use the preprocess function to clear z and d_z
    clearHostAndDeviceArray(z, d_z, size);

    // Start threadPerBlock as 128, then try out differnt configurations: 32, 64, 256, 512, 1024
    // Use divup to get the number of blocks to launch.
    const unsigned threadsPerBlock = 128;

    const unsigned blocks = divup(size, threadsPerBlock);

    // Launch the GPU kernel with blocks and threadPerBlock as launch configuration
    saxpy<<<blocks, threadsPerBlock>>>(d_z, d_x, d_y, a, size);

    // Copy the answer back to the host (CPU) from the device (GPU).
    CUDA(hipMemcpy(z, d_z, count, hipMemcpyDeviceToHost));

    // LOOK: Use postprocess to check the result
    compareReferenceAndResult(z_gold, z, size, 1e-6);
    std::cout << "****************************************************" << std::endl << std::endl;
    ////////////////////////////////////////////////////////////

    // Free device memory using hipFree
    CUDA(hipFree(d_x));
    CUDA(hipFree(d_y));
    CUDA(hipFree(d_z));

    // free host memory
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] z_gold;

    // successful program termination
    return 0;
}
